#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cmath>

using namespace std;

#define check_err(cu_err) { cu_err_handler(cu_err, __FILE__, __LINE__); }

inline void cu_err_handler(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		fprintf(stderr, "GPU error: '%s' at %s:%d", hipGetErrorString(err), file, line);
		exit(1);
	}
}

__global__ void tree_reduction(
		double* tree,
		double stock_price,
		double strike_price,
		int num_steps,
		double R,
		double up_factor,
		double up_prob) {

    for (int step = num_steps-1; step >= 0; --step) {
    	int branch = threadIdx.x;
    	if (branch <= step) {
            double binomial = 1/R * (up_prob * tree[branch + 1] + (1 - up_prob) * tree[branch]);
            double exercise = strike_price - stock_price * pow(up_factor, 2 * branch - step);
            tree[branch] = max(binomial, exercise);
    	}
    	__syncthreads();
    }
}


double binomial_american_put(double stock_price,
                             double strike_price,
                             double expire,
                             double volat,
                             int num_steps,
                             double risk_free_rate) {
    double dt = expire / num_steps;
    double up_factor = exp(volat * sqrt(dt));
    double down_factor = 1 / up_factor;
    double R = exp((risk_free_rate) * dt);
    double up_prob = (R - down_factor) / (up_factor - down_factor);
    double *host_tree = new double[num_steps+1];
    double *dev_tree;

    // Initialize end of host_tree at expire time
    for (int step = 0; step <= num_steps; ++step) {
        // Option value when exercising the option
        double exercise = strike_price - stock_price * pow(up_factor, 2 * step - num_steps);
        host_tree[step] = max(exercise, .0);
    }

    check_err(hipMalloc((void**) &dev_tree, (num_steps+1) * sizeof(double)));
    check_err(hipMemcpy(dev_tree, host_tree, (num_steps+1) * sizeof(double), hipMemcpyHostToDevice));

    tree_reduction<<<1, num_steps>>>(dev_tree, stock_price, strike_price, num_steps, R, up_factor, up_prob);


    double price;
    check_err(hipMemcpy(&price, &dev_tree[0], sizeof(double), hipMemcpyDeviceToHost));

    hipFree(dev_tree);
    delete[] host_tree;
    return price;
}
int main() {
	hipEvent_t start, end;
	check_err(hipEventCreate(&start));
	check_err(hipEventCreate(&end));

    // Warmup
    for (int i = 0; i < 100; ++i) {
        binomial_american_put(20, 25, .5, 1, 200, 0.06);
    }

    check_err(hipEventRecord(start, 0));

    for (int i = 0; i < 1000; ++i) {
        binomial_american_put(20, 25, .5, 1, 200, 0.06);
    }

    check_err(hipEventRecord(end, 0));
    check_err(hipEventSynchronize(end));

    double duration;
    check_err(hipEventElapsedTime(&duration, start, end));
    check_err(hipEventDestroy(start));
    check_err(hipEventDestroy(end));

    printf("Time: %d ms\n", duration);

    return 0;
}
